#include "hip/hip_runtime.h"

#include "utils.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <>
#include <hip/device_functions.h>
#include <thrust/sort.h>

__global__ void histo_kernel(unsigned int * d_out, unsigned int* const d_in,
  unsigned int shift, const unsigned int numElems)
{
  unsigned int mask = 1 << shift;
  if (threadIdx.x + blockDim.x * blockIdx.x >= numElems)  return;
  int bin = (d_in[threadIdx.x + blockDim.x * blockIdx.x] & mask) >> shift;
  atomicAdd(&d_out[bin], 1);
}


__global__ void sumscan_kernel(unsigned int * d_in, const size_t numBins, const unsigned int numElems)
{
  if (threadIdx.x >= numElems)  return;
  extern __shared__ float s_data[];
  s_data[threadIdx.x] = d_in[threadIdx.x];
  __syncthreads(); 
  for (int i = 1; d < numBins; d <<= 2) {
    if (threadIdx.x >= i) {
      s_data[threadIdx.x] += s_data[threadIdx.x - i];
    }
    __syncthreads();
  }
  if (threadIdx.x == 0){
     d_in[0] = 0;
  }
  else{
     d_in[threadIdx.x] = s_data[threadIdx.x- 1];
  }
}

__global__ void makescan_kernel(unsigned int * d_in, unsigned int *d_scan,
  unsigned int shift, const unsigned int numElems)
{
  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  if ( threadIdx.x + blockDim.x * blockIdx.x >= numElems){
     return;
  }
  if(((d_in[myId] & 1 << shift) >> shift)){
 	 d_scan[ threadIdx.x + blockDim.x * blockIdx.x] = 0;
  }
  else{
 	 d_scan[ threadIdx.x + blockDim.x * blockIdx.x] = 1;
  }
}

__global__ void move_kernel(unsigned int* const d_inputVals,
  unsigned int* const d_inputPos,
  unsigned int* const d_outputVals,
  unsigned int* const d_outputPos,
  const unsigned int numElems,
  unsigned int* const d_histogram,
  unsigned int* const d_scaned,
  unsigned int shift)
{
  if (threadIdx.x + blockDim.x * blockIdx.x >= numElems)
     return;
  int desid = 0;
  if ((d_inputVals[threadIdx.x + blockDim.x * blockIdx.x] & 1 << shift;) >> shift) {
    desid = threadIdx.x + blockDim.x * blockIdx.x + d_histogram[1] - d_scaned[threadIdx.x + blockDim.x * blockIdx.x];
  } else {
    desid = d_scaned[threadIdx.x + blockDim.x * blockIdx.x];
  }
  d_outputPos[desid] = d_inputPos[threadIdx.x + blockDim.x * blockIdx.x];
  d_outputVals[desid] = d_inputVals[threadIdx.x + blockDim.x * blockIdx.x];
}

void sort(unsigned int* const d_inputVals,
  unsigned int* const d_inputPos,
  unsigned int* const d_outputVals,
  unsigned int* const d_outputPos,
  const size_t numElems)
{
  const int numBits = 1;  
  const int numBins = 1 << numBits;
  const int m = 1 << 10;
  int blocks = ceil((float)numElems / m);
  printf("m %d blocks %d\n", m ,blocks);
 
  unsigned int *d_binHistogram;
  hipMalloc(&d_binHistogram, sizeof(unsigned int)* numBins);
  thrust::device_vector<unsigned int> d_scan(numElems);

  for (unsigned int i = 0; i < 8 * sizeof(unsigned int); i++) {
    checkCudaErrors(hipMemset(d_binHistogram, 0, sizeof(unsigned int)* numBins));
    histo_kernel << <blocks, m >> >(d_binHistogram, d_inputVals, i, numElems);
    hipDeviceSynchronize();
    sumscan_kernel << <1, numBins, sizeof(unsigned int)* numBins>> >(d_binHistogram, numBins, numElems);
    makescan_kernel << <blocks, m >> >(d_inputVals, thrust::raw_pointer_cast(&d_scan[0]), i, numElems);
    hipDeviceSynchronize();

    thrust::exclusive_scan(d_scan.begin(), d_scan.end(), d_scan.begin());

    hipDeviceSynchronize();
    move_kernel << <blocks, m >> >(d_inputVals, d_inputPos, d_outputVals, d_outputPos,
      numElems, d_binHistogram, thrust::raw_pointer_cast(&d_scan[0]), i);
    hipDeviceSynchronize();


    hipMemcpy(d_inputVals, d_outputVals, numElems * sizeof(unsigned int), hipMemcpyDeviceToDevice);
    hipMemcpy(d_inputPos, d_outputPos, numElems * sizeof(unsigned int), hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();
  }
  
  checkCudaErrors(hipFree(d_binHistogram));
}
